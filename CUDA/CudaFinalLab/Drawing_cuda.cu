#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

// Other
#include <vec3.h>

 // includes, cuda
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Particle
struct Particle 
{
    float3 Position;
    float3 Velocity;
    float3 Colour;
    bool HasCollided;
};

typedef unsigned int uint;
typedef unsigned char uchar;

hipArray* d_imageArray = 0;
const int particle_count = 10000;

__device__ __managed__ uchar4 paper[1000][1000];
__device__ __managed__ Particle particles[particle_count];
__device__ __managed__ int col_count;
__device__ __constant__ float3 gravity{ 0.0f, 9.8f, 0.0f };
__device__ __constant__ float drag = 0.05f;

__device__ void print_vec3(vec3 pVecToPrint)
{
    printf("x = %f, y = %f, z = %f\n", pVecToPrint.x(), pVecToPrint.y(), pVecToPrint.z());
}

__global__ void collision_count(int pRunCount)
{
    int particles = particle_count * pRunCount;
    printf("Collision Count: %d\n", col_count);
    printf("Miss Count: %d\n", particles - col_count);
    printf("Total Particles: %d\n", col_count + (particles - col_count));
    col_count = 0;
}

__global__ void move_particle(float3 pColour, float3 pNozzlePos, float3 pPaperPos, float pRand) 
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = x + y * gridDim.x * blockDim.x;

    // Create particle
    Particle p = Particle();
    p.Colour = pColour;
    p.Position = pNozzlePos;
    p.Velocity = float3(pPaperPos - pNozzlePos);
    p.HasCollided = false;

    hiprandState state;
    hiprand_init(pRand, i, 0, &state);

    // Random numbers for spray
    float randAngle = (hiprand_uniform(&state) - 0.5f) / 10.0f;
    float randX = (hiprand_uniform(&state) - 0.5f) / 10.0f;
    float randY = hiprand_uniform(&state)/ 2.0f;
    float randZ = (hiprand_uniform(&state) - 0.5f) / 10.0f;

    // Rotates the spray vector
    float rotX = (cosf(randAngle) * p.Velocity.x) - (sinf(randAngle) * p.Velocity.z);
    float rotZ = (sinf(randAngle) * p.Velocity.x) + (cosf(randAngle) * p.Velocity.z);

    p.Velocity = float3{ (rotX + randX), (p.Velocity.y + randY), (rotZ + randZ) };

    //printf("%d ", i);
    particles[i] = p;

    float time = 0.016;

    while (particles[i].Position.y >= 0.0f)
    {
        // Movement
        float3 vel = particles[i].Velocity * particles[i].Velocity;
        vel = vel * drag;
        float3 acceleration = gravity - vel;

        acceleration = acceleration * 0.5f;
        float timeSquared = time * time;
        acceleration = acceleration * timeSquared;

        float3 dist = particles[i].Velocity * time;
        dist = dist + acceleration;

        if (dist.y > 0)    
            dist = float3{ dist.x, -dist.y, dist.z };
      
        while (particles[i].Position.y >= pPaperPos.y)
        {
            particles[i].Position += dist;
        }
        particles[i].HasCollided = true;
    }
}

__global__ void collide_particle(float3 pPaperPos)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = x + y * gridDim.x * blockDim.x;
    
    float paperWidth = 1.0f, paperHeight = 1.0f;

    float xLowerBound = -(paperWidth / 2.0f) + pPaperPos.x, 
          xUpperBound = (paperWidth / 2.0f) + pPaperPos.x, 
          zLowerBound = -(paperWidth / 2.0f) + pPaperPos.z, 
          zUpperBound = (paperWidth / 2.0f) + pPaperPos.z;

    while (true)
    {
        if (particles[i].HasCollided)
        {
            // If particle is within the bounds of the paper
            if (particles[i].Position.x >= xLowerBound &&
                particles[i].Position.x <= xUpperBound &&
                particles[i].Position.z >= zLowerBound &&
                particles[i].Position.z <= zUpperBound)
            {
                float pixelPosX = (particles[i].Position.x * 1000.0f), pixelPosZ = (particles[i].Position.z * 1000.0f);
                pixelPosX += (paperWidth / 2.0f) * 1000.0f;
                pixelPosZ += (paperWidth / 2.0f) * 1000.0f;

                uchar4 originalColour = paper[static_cast<int>(pixelPosX)][static_cast<int>(pixelPosZ)];
                float3 paperColour = float3{ static_cast<float>(originalColour.x), static_cast<float>(originalColour.y), static_cast<float>(originalColour.z) };
                paperColour = paperColour * (1.0f - 0.5f);

                float3 newColour = particles[i].Colour * 0.5f;
                newColour = newColour + paperColour;
         
                paper[static_cast<int>(pixelPosX)][static_cast<int>(pixelPosZ)] = make_uchar4(newColour.x, newColour.y, newColour.z, 1.0f);
                atomicAdd(&col_count, 1);
                return;
            }
            return;
        }    
    }
}

__global__ void d_render(uchar4* d_output, uint width, uint height) {
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;

    if ((x < width) && (y < height)) {
       d_output[i] = paper[x][y];
    }
}

__global__ void clear(uint width, uint height)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    uint i = __umul24(y, width) + x;

    paper[x][y] = make_uchar4(0xff, 0xff, 0xff, 0xff);
}


extern "C" void freeTexture() {

    checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height,  dim3 blockSize, dim3 gridSize,
     uchar4 * output) {

    d_render << <gridSize, blockSize >> > (output, width, height);

    getLastCudaError("Drawing Kernel Failed");
}

// Simulate particles
extern "C" void simulate_particles(float3 pColour, float3 pPaperPos, float3 pNozzlePos, float pRand, int pRunCount)
{
    hipDeviceSynchronize();

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Ensure only the movement and collision kernels are timed
    hipEventRecord(start);

    float rnd = 0.0f;
    for (int i = 0; i < pRunCount; i++)
    {
        rnd = rand();
        move_particle << <particle_count/4, 4 >> > (pColour, pNozzlePos, pPaperPos, rnd);
        collide_particle << <particle_count/4, 4 >> > (pPaperPos);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    collision_count << <1, 1 >> > (pRunCount);

    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    printf("Time Elapsed: %f ms\n", ms);

    hipDeviceSynchronize();

    getLastCudaError("Simulation Kernel Failed");
}

extern "C" void clear_paper(int width, int height, dim3 blockSize, dim3 gridSize)
{
    clear << <gridSize, blockSize >> > (width, height);
    getLastCudaError("Paper Clearing");
}

#endif